#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2011  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/forces/pair_full_kernel.cuh>
#include <halmd/mdsim/gpu/forces/pair_trunc_kernel.cuh>
#include <halmd/mdsim/gpu/potentials/pair/parabolic_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/pow.hpp>  // std::pow is not a device function
#include <halmd/utility/tuple.hpp>
#include <halmd/mdsim/forces/trunc/local_r4.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace potentials {
namespace pair {
namespace parabolic_kernel {

/** array of potential parameters for all combinations of particle types */
static texture<float4> param_;
/** squares of potential cutoff radius and energy shift for all combinations of particle types */
static texture<float2> rr_en_cut_;

/**
 * power law interaction potential of a pair of particles.
 *
 * @f[  U(r) = \epsilon (r/\sigma)^{-n} @f]
 */
class parabolic
{
public:
    /**
     * Construct power law potential.
     *
     * Fetch potential parameters from texture cache for particle pair.
     *
     * @param type1 type of first interacting particle
     * @param type2 type of second interacting particle
     */
    HALMD_GPU_ENABLED parabolic(
        unsigned int type1, unsigned int type2
      , unsigned int ntype1, unsigned int ntype2
    )
      : pair_(tex1Dfetch(param_, type1 * ntype2 + type2))
      , pair_rr_en_cut_(tex1Dfetch(rr_en_cut_, type1 * ntype2 + type2))
    {}

    /**
     * Returns square of cutoff distance.
     */
    HALMD_GPU_ENABLED float rr_cut() const
    {
        return pair_rr_en_cut_[0];
    }

    /**
     * Check whether particles are in interaction range.
     *
     * @param rr squared distance between particles
     */
    template <typename float_type>
    HALMD_GPU_ENABLED bool within_range(float_type rr) const
    {
        return (rr < pair_rr_en_cut_[0]);
    }

    /**
     * Compute force and potential for interaction.
     *
     * @param rr squared distance between particles
     * @returns tuple of unit "force" @f$ -U'(r)/r @f$ and potential @f$ U(r) @f$
     *
     * @f{eqnarray*}{
     *   - U'(r) / r &=& n r^{-2} \epsilon (r/\sigma)^{-n} \\
     *   U(r) &=& \epsilon (r/\sigma)^{-n}
     * @f}
     */
    template <typename float_type>
    HALMD_GPU_ENABLED tuple<float_type, float_type> operator()(float_type rr) const
    {
        float_type r = sqrt(rr);
        float_type one_minus_r = 1 - r /  pair_[CUTOFF];
        unsigned short n = static_cast<unsigned short>(pair_[INDEX]);

        float_type real_force = pair_[EPSILON] * halmd::pow(one_minus_r, n - 1);
        float_type fval = real_force / r;

        float_type en_pot = real_force * one_minus_r / n;

        return make_tuple(fval, en_pot);
    }

private:
    /** potential parameters for particle pair */
    fixed_vector<float, 4> pair_;
    /** squared cutoff radius and energy shift for particle pair */
    fixed_vector<float, 2> pair_rr_en_cut_;
};

} // namespace parabolic_kernel

cuda::texture<float4> parabolic_wrapper::param = parabolic_kernel::param_;
cuda::texture<float2> parabolic_wrapper::rr_en_cut = parabolic_kernel::rr_en_cut_;

} // namespace pair
} // namespace potentials

// explicit instantiation of force kernels
namespace forces {

using namespace halmd::mdsim::gpu::potentials::pair::parabolic_kernel;
using namespace halmd::mdsim::forces::trunc;

template class pair_full_wrapper<3, parabolic>;
template class pair_full_wrapper<2, parabolic>;

template class pair_trunc_wrapper<3, parabolic>;
template class pair_trunc_wrapper<2, parabolic>;
template class pair_trunc_wrapper<3, parabolic, local_r4<float> >;
template class pair_trunc_wrapper<2, parabolic, local_r4<float> >;

} // namespace forces

} // namespace gpu
} // namespace mdsim
} // namespace halmd
