#include "hip/hip_runtime.h"
/*
 * Copyright © 2014 Jörg Bartnick
 * Copyright © 2008-2014 Felix Höfling
 * Copyright © 2008-2012 Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// uncomment this line for a thread-divergent, slower implementation
// of the original thermostat introduced by H. C. Andersen (1978).
// #define USE_ORIGINAL_ANDERSEN_THERMOSTAT

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/integrators/brownian_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/random/gpu/normal_distribution.cuh>
#include <halmd/random/gpu/random_number_generator.cuh>
#include <halmd/utility/gpu/thread.cuh>

#if __CUDA_ARCH__ < 120
# define USE_ORIGINAL_ANDERSEN_THERMOSTAT
#endif

namespace halmd {
namespace mdsim {
namespace gpu {
namespace integrators {
namespace brownian_kernel {


/**
 *
 * CUDA execution dimensions must agree with random number generator
 *
 * @param g_velocity particle velocities (array of size \code{} 2 * nplace \endcode for dsfloat arithmetic)
 * @param g_force particle forces (array of size \code{} nplace \endcode)
 * @param timestep integration time-step
 * @param width_rand_force Width of the random force ( = sqrt(2 kT xi / dt) )
 * @param coll_prob collision probability with heat bath
 * @param npart number of particles
 * @param nplace number of placeholder particles
 * @param rng random number generator
 * 
 * 
 */
template <int dimension, typename float_type, typename gpu_vector_type, typename rng_type>
__global__ void integrate(
    float4* g_position
  , gpu_vector_type* g_image
  , float4* g_velocity
  , gpu_vector_type const* g_force
  , float width_rand_force
  , float timestep_by_mobility
  , unsigned int npart
  , unsigned int nplace
  , rng_type rng
  , fixed_vector<float, dimension> box_length
)
{
    typedef fixed_vector<float_type, dimension> vector_type;
    typedef fixed_vector<float, dimension> float_vector_type;
 
    
    // read random number generator state from global device memory
    typename rng_type::state_type state = rng[GTID];

    // cache second normal variate for odd dimensions
    bool cached = false;
    float_type cache;
    
    for (uint i = GTID; i < npart; i += GTDIM) {
        // read position, species, velocity, mass, image, force from global memory
        vector_type r;
        unsigned int species;
        fixed_vector<float_type, dimension> v;
        float mass;

#ifdef USE_VERLET_DSFUN
        tie(r, species) <<= tie(g_position[i], g_position[i + nplace]);
        tie(v, mass) <<= tie(g_velocity[i], g_velocity[i + nplace]);
#else
        tie(r, species) <<= g_position[i];
        tie(v, mass) <<= g_velocity[i];
#endif

        float_vector_type f_rand;
        float_vector_type f = g_force[i];
        f_rand = 0;
        if (width_rand_force != 0) {
            /* Width of the random force = sqrt(2 kT xi / dt) 
             * => random displacement with sigma = sqrt (2 D dt) = sqrt(2 kT dt / xi))
             * the random force is multiplied again by timestep_by_mobility, thus the difference
             */
            // parameters for normal distribution
            float const mean = 0;
            float const sigma = width_rand_force;
            // assign random velocity according to Maxwell-Boltzmann distribution
            for (uint j = 0; j < dimension - 1; j += 2) {
            tie(f_rand[j], f_rand[j + 1]) = normal(rng, state, mean, sigma);
            }
            if (dimension % 2) {
            if ((cached = !cached)) {
                tie(f_rand[dimension - 1], cache) = normal(rng, state, mean, sigma);
            }   
            else {
                f_rand[dimension - 1] = cache;
            }
            }
        }
        // advance position
        v = (f + f_rand);
        r += v * timestep_by_mobility;

        float_vector_type image = box_kernel::reduce_periodic(r, box_length);
        // store position, species, velocity, mass, image in global memory
#ifdef USE_VERLET_DSFUN
        tie(g_position[i], g_position[i + nplace]) <<= tie(r, species);
        tie(g_velocity[i], g_velocity[i + nplace]) <<= tie(v, mass);
#else
        g_position[i] <<= tie(r, species);
        g_velocity[i] <<= tie(v, mass);
#endif
        
        if (!(image == float_vector_type(0))) {
            g_image[i] = image + static_cast<float_vector_type>(g_image[i]);
        }
    } 
    // store random number generator state in global device memory
    rng[GTID] = state; 
} 
} // namespace brownian_kernel

template <int dimension, typename rng_type>
brownian_wrapper<dimension, rng_type> const
brownian_wrapper<dimension, rng_type>::kernel = {
#ifdef USE_VERLET_DSFUN
   brownian_kernel::integrate<dimension, dsfloat>
#else
   brownian_kernel::integrate<dimension, float>
#endif
};

template class brownian_wrapper<3, random::gpu::rand48_rng>;
template class brownian_wrapper<2, random::gpu::rand48_rng>;

} // namespace mdsim
} // namespace gpu
} // namespace integrators
} // namespace halmd
